#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <cstdint>
#include <unistd.h>

__global__
void add(std::size_t n, float *x, float *y) {
  //  int index = threadIdx.x;
  //  int stride = blockDim.x;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  //  printf("Stride %d, index: %d\n", stride, index);
  for (int i = index; i < n; i += stride) {
    y[i] = x[i] + y[i];
    //  printf("i %d  y[i]: %f\n", i, y[i]);
  }
}

int main(int argc, char* argv[]) {

  constexpr std::size_t N = 1<<24;
  std::cout << "N: " << N << std::endl;

  float *x = nullptr;
  float *y = nullptr;

  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (std::size_t i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }
  //  sleep(5);

  // Run kernel on 1M elements on the CPU
  // This threads doesn't need to be compile time constant!
  //  int threads = 256;
  int blockSize = 256;
  int numBlocks = ((N + blockSize - 1) / blockSize);
  //  int blockSize = 1;
  //  int numBlocks = 1;
  add<<<numBlocks, blockSize>>>(N, x, y);

  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (std::size_t i = 0; i < N; i++){
    const auto difference = fabs(y[i]-3.0f);
    if (difference > 0.01) {
      std::cout << "i: " << i << " has: " << difference << std::endl;
    }
    //  printf("i %lu  y[i]: %f\n", i, y[i]);
    maxError = fmax(maxError,  difference);
  }
  std::cout << "Max error: " << maxError << std::endl;
  // Max error is 1.

  std::cout << std::endl;
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0); // 0-th device
  std::cout << "name: " << deviceProp.name << std::endl;
  std::cout << "multiProcessorCount: " << deviceProp.multiProcessorCount << std::endl;
  std::cout << "memoryClockRate: " << deviceProp.memoryClockRate << std::endl;
  std::cout << "maxThreadsPerMultiProcessor: " << deviceProp.maxThreadsPerMultiProcessor << std::endl;
  std::cout << "maxThreadsPerBlock: " << deviceProp.maxThreadsPerBlock << std::endl;
  std::cout << "maxBlocksPerMultiProcessor: " << deviceProp.maxBlocksPerMultiProcessor << std::endl;
  std::cout << std::endl;
  // Free memory
  hipFree(x);
  hipFree(y);
  return 0;
}