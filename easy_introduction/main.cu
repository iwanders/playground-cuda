#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <cstdint>

__global__
void add(std::size_t n, float *x, float *y)
{
  for (std::size_t i = 0; i < n; i++) {
    y[i] = x[i] + y[i];
  }
}

int main(int argc, char* argv[]) {
  constexpr std::size_t N = 1<<20;

  float *x = nullptr;
  float *y = nullptr;

  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (std::size_t i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 1M elements on the CPU
  add<<<1, 1>>>(N, x, y);

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (std::size_t i = 0; i < N; i++){
    maxError = std::max(maxError, std::fabs(y[i]-3.0f));
  }
  std::cout << "Max error: " << maxError << std::endl;
  // Max error is 1.

  std::cout << std::endl;
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0); // 0-th device
  std::cout << "name: " << deviceProp.name << std::endl;
  std::cout << "multiProcessorCount: " << deviceProp.multiProcessorCount << std::endl;
  std::cout << "memoryClockRate: " << deviceProp.memoryClockRate << std::endl;
  std::cout << "maxThreadsPerMultiProcessor: " << deviceProp.maxThreadsPerMultiProcessor << std::endl;
  std::cout << "maxThreadsPerBlock: " << deviceProp.maxThreadsPerBlock << std::endl;
  std::cout << "maxBlocksPerMultiProcessor: " << deviceProp.maxBlocksPerMultiProcessor << std::endl;
  std::cout << std::endl;
  // Free memory
  hipFree(x);
  hipFree(y);
  return 0;
}